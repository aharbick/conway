#include <hip/hip_runtime.h>
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <pthread.h>
#include <unistd.h>
#include <argp.h>

typedef unsigned char ubyte;
typedef unsigned long long ulong64;

__constant__ ulong64 gNeighborFilters[64] = {
  // Row 0 pixels
  (ulong64) 770,
  (ulong64) 1797 << 0,
  (ulong64) 1797 << 1,
  (ulong64) 1797 << 2,
  (ulong64) 1797 << 3,
  (ulong64) 1797 << 4,
  (ulong64) 1797 << 5,
  (ulong64) 49216,

  // Row 1 pixels
  (ulong64) 197123,
  (ulong64) 460039 << 0,
  (ulong64) 460039 << 1,
  (ulong64) 460039 << 2,
  (ulong64) 460039 << 3,
  (ulong64) 460039 << 4,
  (ulong64) 460039 << 5,
  (ulong64) 12599488,

  // Row 2 pixels
  (ulong64) 197123 << 8,
  (ulong64) 460039 << 8 << 0,
  (ulong64) 460039 << 8 << 1,
  (ulong64) 460039 << 8 << 2,
  (ulong64) 460039 << 8 << 3,
  (ulong64) 460039 << 8 << 4,
  (ulong64) 460039 << 8 << 5,
  (ulong64) 12599488 << 8,

  // Row 3 pixels
  (ulong64) 197123 << 16,
  (ulong64) 460039 << 16 << 0,
  (ulong64) 460039 << 16 << 1,
  (ulong64) 460039 << 16 << 2,
  (ulong64) 460039 << 16 << 3,
  (ulong64) 460039 << 16 << 4,
  (ulong64) 460039 << 16 << 5,
  (ulong64) 12599488 << 16,

  // Row 4 pixels
  (ulong64) 197123 << 24,
  (ulong64) 460039 << 24 << 0,
  (ulong64) 460039 << 24 << 1,
  (ulong64) 460039 << 24 << 2,
  (ulong64) 460039 << 24 << 3,
  (ulong64) 460039 << 24 << 4,
  (ulong64) 460039 << 24 << 5,
  (ulong64) 12599488 << 24,

  // Row 5 pixels
  (ulong64) 197123 << 32,
  (ulong64) 460039 << 32 << 0,
  (ulong64) 460039 << 32 << 1,
  (ulong64) 460039 << 32 << 2,
  (ulong64) 460039 << 32 << 3,
  (ulong64) 460039 << 32 << 4,
  (ulong64) 460039 << 32 << 5,
  (ulong64) 12599488 << 32,

  // Row 6 pixels
  (ulong64) 197123 << 40,
  (ulong64) 460039 << 40 << 0,
  (ulong64) 460039 << 40 << 1,
  (ulong64) 460039 << 40 << 2,
  (ulong64) 460039 << 40 << 3,
  (ulong64) 460039 << 40 << 4,
  (ulong64) 460039 << 40 << 5,
  (ulong64) 12599488 << 40,

  // Row 7 pixels
  (ulong64) 515 << 48,
  (ulong64) 1287 << 48 << 0,
  (ulong64) 1287 << 48 << 1,
  (ulong64) 1287 << 48 << 2,
  (ulong64) 1287 << 48 << 3,
  (ulong64) 1287 << 48 << 4,
  (ulong64) 1287 << 48 << 5,
  (ulong64) 16576 << 48
};

__device__ ulong64 computeNextGeneration(ulong64 currentGeneration) {
  ulong64 nextGeneration = currentGeneration;
  for (int i = 0; i < 64; i++) {
    ulong64 neighbors = __popcll(currentGeneration & gNeighborFilters[i]);
    if (currentGeneration & (1UL << i)) {
      // Currently alive...
      if (neighbors <= 1) {
        // DIE - lonely
        nextGeneration &= ~(1UL << i);
      }
      else if (neighbors >= 4) {
        // DIE - too crowded
        nextGeneration &= ~(1UL << i);
      }
    }
    else {
      // Currently dead
      if (neighbors == 3) {
        // BIRTH - perfect number of neighbors
        nextGeneration |= 1UL << i;
      }
    }
  }
  return nextGeneration;
}

__device__ ulong64 countGenerations(ulong64 pattern) {
  // Using a set/map/hash to spot cycles should be faster in general for this
  // problem since the number of generations is relatively small.  However on a
  // CUDA core we don't have easy access to such data structures so instead we
  // use Floyd's algorithm for cycle detection:
  // https://en.wikipedia.org/wiki/Cycle_detection#Floyd's_tortoise_and_hare
  bool ended = false;
  ulong64 generations = 0;
  ulong64 slow = pattern;
  ulong64 fast = computeNextGeneration(slow);
  do {
    generations++;
    ulong64 nextSlow = computeNextGeneration(slow);

    if (slow == nextSlow) {
      ended = true; // If we didn't change then we ended
      break;
    }
    slow = nextSlow;
    fast = computeNextGeneration(computeNextGeneration(fast));
  }
  while (slow != fast);
  ended = slow == 0; // If we died out then we ended

  return ended ? generations : 0;
}

__global__ void evaluateRange(ulong64 beginAt, ulong64 endAt,
                              ulong64 *bestPattern, ulong64 *bestGenerations) {
  for (int pattern = beginAt + (blockIdx.x * blockDim.x + threadIdx.x);
       pattern < endAt;
       pattern += blockDim.x * gridDim.x) {
    ulong64 generations = countGenerations(pattern);
    ulong64 old = atomicMax(bestGenerations, generations);
    if (old < generations) {
      *bestPattern = pattern;
    }
  }
}

void asBinary(ulong64 number, char *buf) {
  for (int i = 63; i >= 0; i--) {
    buf[-i+63] = (number >> i) & 1 ? '1' : '0';
  }
}

const char *prog = "find-cuda-optimal v0.1";
const char *prog_bug_email = "aharbick@aharbick.com";
static char prog_doc[] = "CUDA based exhaustive search for terminal and stable states in an 8x8 bounded Conway's Game of Life grid";
static char prog_args_doc[] = "";
static struct argp_option argp_options[] = {
  { "chunkroot", 'c', "chunkroot", 0, "Process chunks of candidates in 2^chunkroot."},
  { "gpus", 'g', "num", 0, "How many GPUs to use."},
  { "blocksize", 'b', "size", 0, "Size of CUDA block in kernel call."},
  { "threadsperblock", 't', "threads", 0, "Threads per block in CUDA kernel call."},
  { "random", 'r', 0, OPTION_ARG_OPTIONAL, "Search each chunk randomly."},
  { 0 }
};

typedef struct prog_args {
  int threadId = 0;
  bool random = false;
  ulong64 chunkSize = 1024*1024;
  int gpusToUse = 1;
  int blockSize = 4096;
  int threadsPerBlock = 256;
  ulong64 beginAt;
  ulong64 endAt;
} prog_args;

static error_t parse_argp_options(int key, char *arg, struct argp_state *state) {
  prog_args *a = (prog_args *)state->input;
  switch(key) {
  case 'c':
    a->chunkSize = atoi(arg) * atoi(arg);
    break;
  case 'g':
    a->gpusToUse = atoi(arg);
    break;
  case 'b':
    a->blockSize = atoi(arg);
    break;
  case 't':
    a->threadsPerBlock = atoi(arg);
    break;
  default: return ARGP_ERR_UNKNOWN;
  }
  return 0;
}

void *cudaSearch(void *args) {
  prog_args *cli = (prog_args *)args;

  // Choose our GPU
  printf("[Thread %d] %lu - %lu\n", cli->threadId, cli->beginAt, cli->endAt);
  hipSetDevice(cli->threadId);

  // Allocate memory on CUDA device and locally on host to get the best answers
  ulong64 *devBestPattern, *hostBestPattern;
  hostBestPattern = (ulong64 *)malloc(sizeof(ulong64));
  hipMalloc((void**)&devBestPattern, sizeof(ulong64));

  ulong64 *devBestGenerations, *hostBestGenerations;
  hostBestGenerations = (ulong64 *)malloc(sizeof(ulong64));
  *hostBestGenerations = 0;
  hipMalloc((void**)&devBestGenerations, sizeof(ulong64));

  ulong64 chunk = 1;
  ulong64 i = cli->beginAt;
  while (i < cli->endAt) {
    unsigned j = (i+cli->chunkSize) > cli->endAt ? cli->endAt : i+cli->chunkSize;

    hipMemcpy(devBestGenerations, hostBestGenerations, sizeof(ulong64), hipMemcpyHostToDevice);
    evaluateRange<<<cli->blockSize, cli->threadsPerBlock>>>(i, j, devBestPattern, devBestGenerations);

    // Copy device answer to host and emit
    ulong64 prev = *hostBestPattern;
    hipMemcpy(hostBestPattern, devBestPattern, sizeof(ulong64), hipMemcpyDeviceToHost);
    hipMemcpy(hostBestGenerations, devBestGenerations, sizeof(ulong64), hipMemcpyDeviceToHost);
    if (prev != *hostBestPattern) {
      char bin[65] = {'\0'};
      asBinary(*hostBestPattern, bin);
      printf("[Thread %d] %lu generations : %lu :%s\n",
             cli->threadId, *hostBestGenerations, *hostBestPattern, bin);
    }

    if (chunk % 1000 == 0) { // every billion
      printf("[Thread %d] Up to %lu, %2.10f%% complete\n", cli->threadId, i, (float) i/cli->endAt * 100);
    }

    chunk++;
    i += cli->chunkSize;
  }

  return NULL;
}

static struct argp argp = {argp_options, parse_argp_options, prog_args_doc, prog_doc, 0, 0};

int main(int argc, char *argv[]) {
  setvbuf(stdout, NULL, _IONBF, 0);

  // Process the arguments
  prog_args *cli = (prog_args *) malloc(sizeof(prog_args));
  cli->chunkSize = 1024*1024;
  cli->gpusToUse = 1;
  cli->blockSize = 4096;
  cli->threadsPerBlock = 256;
  argp_parse(&argp, argc, argv, 0, 0, cli);

  // We're going to spin up one CPU thread per GPU and assign that an equal portion of the search space
  ulong64 candidatesPerGpu = ULONG_MAX / cli->gpusToUse;
  pthread_t *threads = (pthread_t *) malloc(sizeof(pthread_t) * cli->gpusToUse);

  for (int t = 0; t < cli->gpusToUse; t++) {
    // Spin up a thread per gpu
    prog_args *targs = (prog_args *) malloc(sizeof(prog_args));
    memcpy(targs, cli, sizeof(prog_args));
    targs->threadId = t;
    targs->beginAt = t * candidatesPerGpu + 1;
    targs->endAt = targs->beginAt + candidatesPerGpu -1;
    pthread_create(&threads[t], NULL, cudaSearch, (void*) targs);
    sleep(5);
  }

  for (int t = 0; t < cli->gpusToUse; t++) {
    pthread_join(threads[t], NULL);
    printf("[Thread %d] COMPLETE\n", t);
  }

  return 0;
}
